#include "hip/hip_runtime.h"
#include "Particles.cuh"

#include <random>
#include <chrono>
#include <cudaDefs.h>

Particles::Particles(shared_ptr<Settings> settings, shared_ptr<HeightMap> hMap) {
    this->settings = settings;
    this->hMap = hMap;

    this->activeLeaders = settings->leaders;
    this->activeFollowers = settings->followers;

    float w = float(hMap->glData.imageWidth);
    float h = float(hMap->glData.imageHeight);
    this->generateOnGPU(settings->leaders, w, h, this->dLeaderPos, this->dLeaderVel);
    this->generateOnGPU(settings->followers, w, h, this->dFollowerPos, this->dFollowerVel);

    this->status = hipblasStatus_t();
    this->handle = hipblasHandle_t();
    this->status = hipblasCreate(&handle) ;

    unsigned int onesCount = 2 * (settings->leaders > settings->followers ? settings->leaders : settings->followers);
    hipMalloc((void**)&this->dOnes, sizeof(float) * onesCount);
    hipMemcpy(dOnes, vector<float>(onesCount, 1.0f).data(), sizeof(float) * onesCount, hipMemcpyHostToDevice);

    //hipMalloc( (void**)&dDistances, settings->leaders * settings->followers * sizeof(float));
}

Particles::~Particles() {
    status = hipblasDestroy(handle);
    if (this->dLeaderPos) hipFree(this->dLeaderPos);
    if (this->dLeaderVel) hipFree(this->dLeaderVel);
    if (this->dFollowerPos) hipFree(this->dFollowerPos);
    if (this->dFollowerVel) hipFree(this->dFollowerVel);
}

pair<vector<float2>, vector<float2>> Particles::generate(int n, float imgWidth, float imgHeight) {
    vector<float2> pos;
    vector<float2> vel;

    std::mt19937 generator(std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count());
    std::uniform_real_distribution<float> dis(0.0, 1.0);

    float dW = imgWidth / float(n);
    float dH = imgHeight / float(n);
    for (int i = 0; i < n; i++) {
        pos.push_back(float2 {i * dW, i * dH});
        //pos.push_back(float2 {dis(generator) * imgWidth, dis(generator) * imgWidth});
        vel.push_back(float2 {0, 0});
    }


    return make_pair(pos, vel);
}

void Particles::generateOnGPU(int n, float imgWidth, float imgHeight, float2* &pos, float2* &vel) {
    checkCudaErrors(hipMalloc((void**)&pos, n * sizeof(float2)));
    checkCudaErrors(hipMalloc((void**)&vel, n * sizeof(float2)));
    auto tmp = this->generate(n, imgWidth, imgHeight);
    checkCudaErrors(hipMemcpy(pos, tmp.first.data(), n * sizeof(float2), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(vel, tmp.second.data(), n * sizeof(float2), hipMemcpyHostToDevice));
}


__global__ void clearPBO(unsigned char* pbo, const unsigned int pboWidth, const unsigned int pboHeight) {
    unsigned int tx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int ty = blockIdx.y * blockDim.y + threadIdx.y;

    if (tx >= pboWidth || ty > pboHeight) return;
    unsigned int pboIdx = ((ty * pboWidth) + tx) * 4;

    pbo[pboIdx++] = 0;
    pbo[pboIdx++] = 0;
    pbo[pboIdx++] = 0;
    pbo[pboIdx]   = 0;
}

__global__ void renderParticles(uchar3 color, float2* particles, int particleCount, unsigned char* pbo, const unsigned int pboWidth, const unsigned int pboHeight) {
    unsigned int tx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int jump = blockDim.x * gridDim.x;

    while (tx < particleCount) {
        float2 p = particles[tx];
        unsigned int pboIdx = ((floor(p.y) * pboWidth) + floor(p.x)) * 4;
        pbo[pboIdx++] = color.x;
        pbo[pboIdx++] = color.y;
        pbo[pboIdx++] = color.z;
        pbo[pboIdx]   = 255;

        tx += jump;
    }
}

void Particles::renderToOverlay() {
    checkCudaErrors(hipGraphicsMapResources(1, &hMap->cudaData.pboResource, 0));
    unsigned char* pboData;
    size_t pboSize;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&pboData, &pboSize, hMap->cudaData.pboResource));

    {   // CLEAR PBO
        constexpr unsigned int TPB_1D = 8; //TODO -> define somewhere TPB_1D
        dim3 block(TPB_1D, TPB_1D, 1);
        dim3 grid((hMap->glData.imageWidth + TPB_1D - 1) / TPB_1D, (hMap->glData.imageHeight + TPB_1D - 1) / TPB_1D, 1);
        clearPBO<<<grid, block>>>(pboData, hMap->glData.imageWidth, hMap->glData.imageHeight);
    };

    {   // PUT PARTCLES INTO PBO
        //TODO -> adjust block and grid sizes
        constexpr unsigned int TPB_1D = 128; //TODO -> define somewhere TPB_1D
        dim3 block(128, 1, 1);
        dim3 grid((hMap->glData.imageWidth + TPB_1D - 1) / TPB_1D, 1, 1);
        renderParticles<<<grid, block>>>(Particles::leaderColor,    dLeaderPos,   settings->leaders,   pboData, hMap->glData.imageWidth, hMap->glData.imageHeight);
        renderParticles<<<grid, block>>>(Particles::followerColor,  dFollowerPos, settings->followers, pboData, hMap->glData.imageWidth, hMap->glData.imageHeight);
    };

    checkCudaErrors(hipGraphicsUnmapResources(1, &hMap->cudaData.pboResource, 0));

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, hMap->glData.pboID);
    auto err = glGetError();
    glBindTexture(GL_TEXTURE_2D, hMap->overlayTexId);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, hMap->glData.imageWidth, hMap->glData.imageHeight, GL_RGBA, GL_UNSIGNED_BYTE, NULL);   //Source parameter is NULL, Data is coming from a PBO, not host memory
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
}


__global__ void createSquareMatrix(float2* particles, int particleCount, float* dst) {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int jump = gridDim.x * blockDim.x;

    float2 * particle = &particles[idx];
    float* row = &dst[2*idx];
    while (idx < particleCount) {
        row[0] = particle->x * particle->x;
        row[1] = particle->y * particle->y;

        particle += jump;
        row += jump;
        idx += jump;
    }
}

inline void squarePositions(float* &dPositionsSq, float2* dParticles, unsigned int activeParticles) {
    checkCudaErrors(hipMalloc((void**)&dPositionsSq, 2 * activeParticles * sizeof(float)));
    constexpr unsigned int TPB_1D = 128;
    dim3 block(128,1,1); //TODO -> change TPB1D?
    dim3 grid((activeParticles + TPB_1D - 1) / TPB_1D);
    createSquareMatrix<<<block, grid>>>(dParticles, activeParticles, dPositionsSq);
}

void Particles::calculateDistances() {
    // M*N distance matrix
    float* dDistances;
    checkCudaErrors(hipMalloc( (void**)&dDistances, activeLeaders * activeFollowers * sizeof(float)));

    // SQUARE OF POSITIONS
    float* dLeadersPosSq;
    float* dFollowersPosSq;
    squarePositions(dLeadersPosSq, dLeaderPos, activeLeaders);
    squarePositions(dFollowersPosSq, dFollowerPos, activeFollowers);

    float alpha = 1.0f;
    float beta = 0.0f;
    auto response = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                activeFollowers, activeLeaders, 2,  //  M, N, K
                                &alpha,
                                dFollowersPosSq, 2,
                                dOnes, 2,
                                &beta,
                                dDistances, activeFollowers);


    //alpha = 1.0f;
    beta = 1.0f;
    response = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                           activeFollowers, activeLeaders, 2,  //  M, N, K
                           &alpha,
                           dOnes, 2,
                           dLeadersPosSq, 2,
                           &beta,
                           dDistances, activeFollowers);

    alpha = -2.0f;
    //beta = 1.0f;
    response = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                activeFollowers, activeLeaders, 2,  //  M, N, K
                                &alpha,
                                (float*)dFollowerPos, 2,
                                (float*)dLeaderPos, 2,
                                &beta,
                                dDistances, activeFollowers);
    checkDeviceMatrix<float>(dDistances,sizeof(float) * activeFollowers, activeLeaders, activeFollowers, "%f ", "M");

    //checkDeviceMatrix<float>(dDistances,sizeof(float2) * activeLeaders, activeFollowers, activeLeaders, "%f ", "M");
    checkDeviceMatrix<float>(dFollowersPosSq,   sizeof(float) * 2, activeFollowers,    2, "%f ", "Follower position - square");
    //checkDeviceMatrix<float>(dLeadersPosSq,     sizeof(float) * 2 * activeLeaders,     activeLeaders,      2, "%f ", "Leader position - square");
    //checkDeviceMatrix<float>(dOnes, sizeof(float)*2, activeLeaders, 2, "%f ", "Ones");
    //checkDeviceMatrix<float>(dDistances,	sizeof(float)*activeFollowers,activeLeaders, activeFollowers, "%f ", "M");
    //checkDeviceMatrix<float>(dFollowersPosSq, sizeof(float)*2, activeFollowers, 2, "%f ", "Leaders pos square");

}








