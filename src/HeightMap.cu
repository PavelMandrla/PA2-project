#include "HeightMap.cuh"

#include <cudaDefs.h>
#include <cuda_gl_interop.h>
#include <GL/freeglut.h>
#include <GL/glew.h>
#include <imageManager.h>
#include <vector>

HeightMap::HeightMap(shared_ptr<Settings> settings) {
    this->settings = settings;
    prepareHeightMapTexture(settings->heightMap.c_str());
    initOverlayTexture();
    initCUDAObjects();
}

HeightMap::~HeightMap() {
    //TODO -> fix Unregistering and unregister overlay texture
//    checkCudaErrors(hipGraphicsUnregisterResource(this->cudaData.texResource));
//    checkCudaErrors(hipGraphicsUnregisterResource(this->cudaData.pboResource));

    if (this->glData.textureID > 0)
        glDeleteTextures(1, &this->glData.textureID);
    if (this->overlayPboID > 0)
        glDeleteBuffers(1, &this->overlayPboID);
}

void HeightMap::prepareHeightMapTexture(const char *imageFileName) {
    FIBITMAP* tmp = ImageManager::GenericLoader(imageFileName, 0);
    glData.imageWidth = FreeImage_GetWidth(tmp);
    glData.imageHeight = FreeImage_GetHeight(tmp);

    //OpenGL Texture
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &glData.textureID);
    glBindTexture(GL_TEXTURE_2D, glData.textureID);

    //WARNING: Just some of inner format are supported by CUDA!!!
    glTexImage2D(GL_TEXTURE_2D, 0, GL_LUMINANCE, glData.imageWidth, glData.imageHeight, 0, GL_RED, GL_UNSIGNED_BYTE, FreeImage_GetBits(tmp));
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);

    glBindTexture(GL_TEXTURE_2D, 0);
    FreeImage_Unload(tmp);
}

void HeightMap::initCUDAObjects() {
    // Register Image to cuda tex resource
    checkCudaErrors(hipGraphicsGLRegisterImage(
            &cudaData.texResource,
            glData.textureID,
            GL_TEXTURE_2D,
            hipGraphicsRegisterFlags::hipGraphicsRegisterFlagsReadOnly
    ));

    // Map reousrce and retrieve pointer to undelying array data
    checkCudaErrors(hipGraphicsMapResources(1, &cudaData.texResource, 0)); //OPENGL, pls nepracuj ted s tou texturou
    checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&cudaData.texArrayData, cudaData.texResource, 0, 0));    //z resourcu chci tahat pixelova data

    // Set resource descriptor
    cudaData.resDesc.resType = hipResourceType::hipResourceTypeArray;
    cudaData.resDesc.res.array.array = cudaData.texArrayData;

    // Set Texture Descriptor: Tex Units will know how to read the texture
    cudaData.texDesc.readMode = hipReadModeElementType;
    cudaData.texDesc.normalizedCoords = false;
    cudaData.texDesc.filterMode = hipFilterModePoint;
    cudaData.texDesc.addressMode[0] = hipAddressModeClamp;
    cudaData.texDesc.addressMode[1] = hipAddressModeClamp;

    // Set Channel Descriptor: How to interpret individual bytes
    checkCudaErrors(hipGetChannelDesc(&cudaData.texChannelDesc, cudaData.texArrayData));

    // Create CUDA Texture Object
    checkCudaErrors(hipCreateTextureObject(&cudaData.texObj, &cudaData.resDesc, &cudaData.texDesc, nullptr));

    // Unmap resource: Release the resource for OpenGL
    checkCudaErrors(hipGraphicsUnmapResources(1, &cudaData.texResource, 0));

    // Register PBO
    checkCudaErrors(hipGraphicsGLRegisterBuffer(
            &cudaData.pboResource,
            this->overlayPboID,
            hipGraphicsRegisterFlags::hipGraphicsRegisterFlagsWriteDiscard
    ));
}

void HeightMap::initOverlayTexture() {
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &overlayTexId);
    glBindTexture(GL_TEXTURE_2D, overlayTexId);

    std::vector<GLubyte> emptyData(settings->heightmapGridX * settings->heightmapGridY * 4, 0);
    glTexImage2D(GL_TEXTURE_2D,0, GL_RGBA, settings->heightmapGridX, settings->heightmapGridY, 0, GL_BGRA, GL_UNSIGNED_BYTE, &emptyData[0]);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    glBindTexture(GL_TEXTURE_2D, 0);

    glGenBuffers(1, &this->overlayPboID);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, this->overlayPboID);														// Make this the current UNPACK buffer (OpenGL is state-based)
    glBufferData(GL_PIXEL_UNPACK_BUFFER, settings->heightmapGridX * settings->heightmapGridY * 4, NULL, GL_DYNAMIC_COPY);	// Allocate data for the buffer. 4-channel 8-bit image
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
}

void HeightMap::display() {
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    glEnable(GL_TEXTURE_2D);

    //glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, this->glData.textureID);
    glBegin(GL_QUADS);
    glTexCoord2d(0, 0);		glVertex2d(0, 0);
    glTexCoord2d(1, 0);		glVertex2d(settings->viewportWidth, 0);
    glTexCoord2d(1, 1);		glVertex2d(settings->viewportWidth, settings->viewportHeight);
    glTexCoord2d(0, 1);		glVertex2d(0, settings->viewportHeight);
    glEnd();

    //glActiveTexture(GL_TEXTURE1);
    glBindTexture(GL_TEXTURE_2D, this->overlayTexId);
    glBegin(GL_QUADS);
    glTexCoord2d(0, 0);		glVertex2d(0, 0);
    glTexCoord2d(1, 0);		glVertex2d(settings->viewportWidth, 0);
    glTexCoord2d(1, 1);		glVertex2d(settings->viewportWidth, settings->viewportHeight);
    glTexCoord2d(0, 1);		glVertex2d(0, settings->viewportHeight);
    glEnd();

    glDisable(GL_TEXTURE_2D);

    glFlush();
    glutSwapBuffers();
}

void HeightMap::resize(GLsizei w, GLsizei h) {
    settings->viewportWidth = w;
    settings->viewportHeight = h;

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    glViewport(0, 0, settings->viewportWidth, settings->viewportHeight);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0, settings->viewportWidth, 0, settings->viewportHeight);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glutPostRedisplay();
}


